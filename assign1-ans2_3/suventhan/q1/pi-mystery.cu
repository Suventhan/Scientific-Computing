#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define TRIALS_PER_THREAD 4096
#define NUM_BLOCK  256  // Number of thread blocks
#define NUM_THREAD  256  // Number of threads per block
#define NBIN 268435456

// #define NBIN TRIALS_PER_THREAD*NUM_THREAD*NUM_BLOCK  // Number of bins 4096*256*256
//
// // 2^28
#define NBIN 268435456  // Number of bins 4096*256*256
// // 2^26
// #define NBIN 67108864  // Number of bins 1024*256*256
// 2^24
// #define NBIN 16777216  // Number of bins 256*256*256
int tid;

#ifdef DP
typedef double Real;
	#define PI  3.14159265358979323846  // known value of pi
#else
typedef float Real;
	#define PI 3.1415926535  // known value of pi
#endif

Real pi_gpu = 0;
Real pi_cpu = 0;

// Kernel that executes on the CUDA device
__global__ void cal_pi(Real *sum, int nbin, Real step, int nthreads, int nblocks) {
	int i;
	Real x;
	int idx = blockIdx.x*blockDim.x+threadIdx.x;  // Sequential thread index across the blocks
	for (i=idx; i< nbin; i+=nthreads*nblocks) {
		x = (i+0.5)*step;
		sum[idx] += 4.0/(1.0+x*x);
	}
}

// Main routine that executes on the host
int main(void) {
	clock_t start, stop;
	dim3 dimGrid(NUM_BLOCK,1,1);  // Grid dimensions
	dim3 dimBlock(NUM_THREAD,1,1);  // Block dimensions
	Real *sumHost, *sumDev;  // Pointer to host & device arrays

	printf("# of trials per thread = %d, # of blocks = %d, # of threads/block = %d\n",TRIALS_PER_THREAD,NUM_BLOCK,NUM_THREAD);

	Real step = 1.0/NBIN;  // Step size
	size_t size = NUM_BLOCK*NUM_THREAD*sizeof(Real);  //Array memory size
	sumHost = (Real *)malloc(size);  //  Allocate array on host

	start = clock();

	hipMalloc((void **) &sumDev, size);  // Allocate array on device
	// Initialize array in device to 0
	hipMemset(sumDev, 0, size);
	// Do calculation on device
	cal_pi <<<dimGrid, dimBlock>>> (sumDev, NBIN, step, NUM_THREAD, NUM_BLOCK); // call CUDA kernel
	// Retrieve result from device and store it in host array
	hipMemcpy(sumHost, sumDev, size, hipMemcpyDeviceToHost);
	for(tid = 0; tid < NUM_THREAD*NUM_BLOCK; tid++)
		pi_gpu += sumHost[tid];
	pi_gpu *= step;

	stop = clock();
	// Print results
	printf("GPU pi calculated in %f s.\n", (stop-start)/(float)CLOCKS_PER_SEC);
	printf("CPU pi calculated in %f s.\n", (stop-start)/(float)CLOCKS_PER_SEC);
	#ifdef DP
	printf("GPU estimate of PI = %20.18f [error of %20.18f]\n", pi_gpu, pi_gpu - PI);
	#else
	printf("GPU estimate of PI = %f [error of %f]\n", pi_gpu, pi_gpu - PI);
	#endif	
	// Cleanup
	free(sumHost);
	hipFree(sumDev);

	return 0;
}
